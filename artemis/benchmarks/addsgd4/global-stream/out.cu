#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 16
#define by 32
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void addsg (double * __restrict__ d_up_0, double * __restrict__ d_up_1, double * __restrict__ d_up_2, double * __restrict__ d_u_0, double * __restrict__ d_u_1, double * __restrict__ d_u_2, double * __restrict__ d_um_0, double * __restrict__ d_um_1, double * __restrict__ d_um_2, double * __restrict__ d_rho, double * __restrict__ strx, double * __restrict__ stry, double * __restrict__ strz, double * __restrict__ dcx, double * __restrict__ dcy, double * __restrict__ dcz, double * __restrict__ cox, double * __restrict__ coy, double * __restrict__ coz, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);

	double (*up_0)[MM][NN] = (double (*)[MM][NN]) d_up_0;
	double (*up_1)[MM][NN] = (double (*)[MM][NN]) d_up_1;
	double (*up_2)[MM][NN] = (double (*)[MM][NN]) d_up_2;
	double (*u_0)[MM][NN] = (double (*)[MM][NN]) d_u_0;
	double (*u_1)[MM][NN] = (double (*)[MM][NN]) d_u_1;
	double (*u_2)[MM][NN] = (double (*)[MM][NN]) d_u_2;
	double (*um_0)[MM][NN] = (double (*)[MM][NN]) d_um_0;
	double (*um_1)[MM][NN] = (double (*)[MM][NN]) d_um_1;
	double (*um_2)[MM][NN] = (double (*)[MM][NN]) d_um_2;
	double (*rho)[MM][NN] = (double (*)[MM][NN]) d_rho;

	//Array and variable declarations
	double birho;
	double beta = 0.625;

	#pragma unroll 4
	for (int k=2; k<=L-3; k++) {
		if (j >= max(2, j0) && j <= min(j0+blockDim.y-1, M-3) && i >= max(2, i0) && i <= min(i0+blockDim.x-1, N-3)) {
			birho = beta / rho[k][j][i];
			up_0[max (0, k)][j][i] -= birho * (strx[i] * coy[j] * coz[k] * (rho[k][j][i+1] * dcx[i+1] * (u_0[k][j][i+2] - 2 * u_0[k][j][i+1] + u_0[k][j][i]) - 2 * rho[k][j][i] * dcx[i] * (u_0[k][j][i+1] - 2 * u_0[k][j][i] + u_0[k][j][i-1]) + rho[k][j][i-1] * dcx[i-1] * (u_0[k][j][i] - 2 * u_0[k][j][i-1] + u_0[k][j][i-2]) - rho[k][j][i+1] * dcx[i+1] * (um_0[k][j][i+2] - 2 * um_0[k][j][i+1] + um_0[k][j][i]) + 2 * rho[k][j][i] * dcx[i] * (um_0[k][j][i+1] - 2 * um_0[k][j][i] + um_0[k][j][i-1]) - rho[k][j][i-1] * dcx[i-1] * (um_0[k][j][i] - 2 * um_0[k][j][i-1] + um_0[k][j][i-2])) + stry[j] * cox[i] * coz[k] * (rho[k][j+1][i] * dcy[j+1] * (u_0[k][j+2][i] - 2 * u_0[k][j+1][i] + u_0[k][j][i]) - 2 * rho[k][j][i] * dcy[j] * (u_0[k][j+1][i] - 2 * u_0[k][j][i] + u_0[k][j-1][i]) + rho[k][j-1][i] * dcy[j-1] * (u_0[k][j][i] - 2 * u_0[k][j-1][i] + u_0[k][j-2][i]) - rho[k][j+1][i] * dcy[j+1] * (um_0[k][j+2][i] - 2 * um_0[k][j+1][i] + um_0[k][j][i]) + 2 * rho[k][j][i] * dcy[j] * (um_0[k][j+1][i] - 2 * um_0[k][j][i] + um_0[k][j-1][i]) - rho[k][j-1][i] * dcy[j-1] * (um_0[k][j][i] - 2 * um_0[k][j-1][i] + um_0[k][j-2][i])) + strz[k] * cox[i] * coy[j] * (rho[k+1][j][i] * dcz[k+1] * (u_0[k+2][j][i] - 2 * u_0[k+1][j][i] + u_0[k][j][i]) - 2 * rho[k][j][i] * dcz[k] * (u_0[k+1][j][i] - 2 * u_0[k][j][i] + u_0[k-1][j][i]) + rho[k-1][j][i] * dcz[k-1] * (u_0[k][j][i] - 2 * u_0[k-1][j][i] + u_0[k-2][j][i]) - rho[k+1][j][i] * dcz[k+1] * (um_0[k+2][j][i] - 2 * um_0[k+1][j][i] + um_0[k][j][i]) + 2 * rho[k][j][i] * dcz[k] * (um_0[k+1][j][i] - 2 * um_0[k][j][i] + um_0[k-1][j][i]) - rho[k-1][j][i] * dcz[k-1] * (um_0[k][j][i] - 2 * um_0[k-1][j][i] + um_0[k-2][j][i])));
			up_1[max (0, k)][j][i] -= birho * (strx[i] * coy[j] * coz[k] * (rho[k][j][i+1] * dcx[i+1] * (u_1[k][j][i+2] - 2 * u_1[k][j][i+1] + u_1[k][j][i]) - 2 * rho[k][j][i] * dcx[i] * (u_1[k][j][i+1] - 2 * u_1[k][j][i] + u_1[k][j][i-1]) + rho[k][j][i-1] * dcx[i-1] * (u_1[k][j][i] - 2 * u_1[k][j][i-1] + u_1[k][j][i-2]) - rho[k][j][i+1] * dcx[i+1] * (um_1[k][j][i+2] - 2 * um_1[k][j][i+1] + um_1[k][j][i]) + 2 * rho[k][j][i] * dcx[i] * (um_1[k][j][i+1] - 2 * um_1[k][j][i] + um_1[k][j][i-1]) - rho[k][j][i-1] * dcx[i-1] * (um_1[k][j][i] - 2 * um_1[k][j][i-1] + um_1[k][j][i-2])) + stry[j] * cox[i] * coz[k] * (rho[k][j+1][i] * dcy[j+1] * (u_1[k][j+2][i] - 2 * u_1[k][j+1][i] + u_1[k][j][i]) - 2 * rho[k][j][i] * dcy[j] * (u_1[k][j+1][i] - 2 * u_1[k][j][i] + u_1[k][j-1][i]) + rho[k][j-1][i] * dcy[j-1] * (u_1[k][j][i] - 2 * u_1[k][j-1][i] + u_1[k][j-2][i]) - rho[k][j+1][i] * dcy[j+1] * (um_1[k][j+2][i] - 2 * um_1[k][j+1][i] + um_1[k][j][i]) + 2 * rho[k][j][i] * dcy[j] * (um_1[k][j+1][i] - 2 * um_1[k][j][i] + um_1[k][j-1][i]) - rho[k][j-1][i] * dcy[j-1] * (um_1[k][j][i] - 2 * um_1[k][j-1][i] + um_1[k][j-2][i])) + strz[k] * cox[i] * coy[j] * (rho[k+1][j][i] * dcz[k+1] * (u_1[k+2][j][i] - 2 * u_1[k+1][j][i] + u_1[k][j][i]) - 2 * rho[k][j][i] * dcz[k] * (u_1[k+1][j][i] - 2 * u_1[k][j][i] + u_1[k-1][j][i]) + rho[k-1][j][i] * dcz[k-1] * (u_1[k][j][i] - 2 * u_1[k-1][j][i] + u_1[k-2][j][i]) - rho[k+1][j][i] * dcz[k+1] * (um_1[k+2][j][i] - 2 * um_1[k+1][j][i] + um_1[k][j][i]) + 2 * rho[k][j][i] * dcz[k] * (um_1[k+1][j][i] - 2 * um_1[k][j][i] + um_1[k-1][j][i]) - rho[k-1][j][i] * dcz[k-1] * (um_1[k][j][i] - 2 * um_1[k-1][j][i] + um_1[k-2][j][i])));
			up_2[max (0, k)][j][i] -= birho * (strx[i] * coy[j] * coz[k] * (rho[k][j][i+1] * dcx[i+1] * (u_2[k][j][i+2] - 2 * u_2[k][j][i+1] + u_2[k][j][i]) - 2 * rho[k][j][i] * dcx[i] * (u_2[k][j][i+1] - 2 * u_2[k][j][i] + u_2[k][j][i-1]) + rho[k][j][i-1] * dcx[i-1] * (u_2[k][j][i] - 2 * u_2[k][j][i-1] + u_2[k][j][i-2]) - rho[k][j][i+1] * dcx[i+1] * (um_2[k][j][i+2] - 2 * um_2[k][j][i+1] + um_2[k][j][i]) + 2 * rho[k][j][i] * dcx[i] * (um_2[k][j][i+1] - 2 * um_2[k][j][i] + um_2[k][j][i-1]) - rho[k][j][i-1] * dcx[i-1] * (um_2[k][j][i] - 2 * um_2[k][j][i-1] + um_2[k][j][i-2])) + stry[j] * cox[i] * coz[k] * (rho[k][j+1][i] * dcy[j+1] * (u_2[k][j+2][i] - 2 * u_2[k][j+1][i] + u_2[k][j][i]) - 2 * rho[k][j][i] * dcy[j] * (u_2[k][j+1][i] - 2 * u_2[k][j][i] + u_2[k][j-1][i]) + rho[k][j-1][i] * dcy[j-1] * (u_2[k][j][i] - 2 * u_2[k][j-1][i] + u_2[k][j-2][i]) - rho[k][j+1][i] * dcy[j+1] * (um_2[k][j+2][i] - 2 * um_2[k][j+1][i] + um_2[k][j][i]) + 2 * rho[k][j][i] * dcy[j] * (um_2[k][j+1][i] - 2 * um_2[k][j][i] + um_2[k][j-1][i]) - rho[k][j-1][i] * dcy[j-1] * (um_2[k][j][i] - 2 * um_2[k][j-1][i] + um_2[k][j-2][i])) + strz[k] * cox[i] * coy[j] * (rho[k+1][j][i] * dcz[k+1] * (u_2[k+2][j][i] - 2 * u_2[k+1][j][i] + u_2[k][j][i]) - 2 * rho[k][j][i] * dcz[k] * (u_2[k+1][j][i] - 2 * u_2[k][j][i] + u_2[k-1][j][i]) + rho[k-1][j][i] * dcz[k-1] * (u_2[k][j][i] - 2 * u_2[k-1][j][i] + u_2[k-2][j][i]) - rho[k+1][j][i] * dcz[k+1] * (um_2[k+2][j][i] - 2 * um_2[k+1][j][i] + um_2[k][j][i]) + 2 * rho[k][j][i] * dcz[k] * (um_2[k+1][j][i] - 2 * um_2[k][j][i] + um_2[k-1][j][i]) - rho[k-1][j][i] * dcz[k-1] * (um_2[k][j][i] - 2 * um_2[k-1][j][i] + um_2[k-2][j][i])));
		}
	}
}


extern "C" void host_code (double *h_up_0, double *h_up_1, double *h_up_2, double *h_u_0, double *h_u_1, double *h_u_2, double *h_um_0, double *h_um_1, double *h_um_2, double *h_rho, double *h_strx, double *h_stry, double *h_strz, double *h_dcx, double *h_dcy, double *h_dcz, double *h_cox, double *h_coy, double *h_coz, int L, int M, int N) {
	double *up_0;
	hipMalloc (&up_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for up_0\n");
	hipMemcpy (up_0, h_up_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *up_1;
	hipMalloc (&up_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for up_1\n");
	hipMemcpy (up_1, h_up_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *up_2;
	hipMalloc (&up_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for up_2\n");
	hipMemcpy (up_2, h_up_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *u_0;
	hipMalloc (&u_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *u_1;
	hipMalloc (&u_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *u_2;
	hipMalloc (&u_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *um_0;
	hipMalloc (&um_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for um_0\n");
	hipMemcpy (um_0, h_um_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *um_1;
	hipMalloc (&um_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for um_1\n");
	hipMemcpy (um_1, h_um_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *um_2;
	hipMalloc (&um_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for um_2\n");
	hipMemcpy (um_2, h_um_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *rho;
	hipMalloc (&rho, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for rho\n");
	hipMemcpy (rho, h_rho, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double )*(N - 0));
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double )*(N - 0), hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double )*(M - 0));
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double )*(M - 0), hipMemcpyHostToDevice);
	double *strz;
	hipMalloc (&strz, sizeof(double )*(L - 0));
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(double )*(L - 0), hipMemcpyHostToDevice);
	double *dcx;
	hipMalloc (&dcx, sizeof(double )*(N - 0));
	check_error ("Failed to allocate device memory for dcx\n");
	hipMemcpy (dcx, h_dcx, sizeof(double )*(N - 0), hipMemcpyHostToDevice);
	double *dcy;
	hipMalloc (&dcy, sizeof(double )*(M - 0));
	check_error ("Failed to allocate device memory for dcy\n");
	hipMemcpy (dcy, h_dcy, sizeof(double )*(M - 0), hipMemcpyHostToDevice);
	double *dcz;
	hipMalloc (&dcz, sizeof(double )*(L - 0));
	check_error ("Failed to allocate device memory for dcz\n");
	hipMemcpy (dcz, h_dcz, sizeof(double )*(L - 0), hipMemcpyHostToDevice);
	double *cox;
	hipMalloc (&cox, sizeof(double )*(N - 0));
	check_error ("Failed to allocate device memory for cox\n");
	hipMemcpy (cox, h_cox, sizeof(double )*(N - 0), hipMemcpyHostToDevice);
	double *coy;
	hipMalloc (&coy, sizeof(double )*(M - 0));
	check_error ("Failed to allocate device memory for coy\n");
	hipMemcpy (coy, h_coy, sizeof(double )*(M - 0), hipMemcpyHostToDevice);
	double *coz;
	hipMalloc (&coz, sizeof(double )*(L - 0));
	check_error ("Failed to allocate device memory for coz\n");
	hipMemcpy (coz, h_coz, sizeof(double )*(L - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, 1);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, blockconfig_1.y), 1);

	addsg <<<gridconfig_1, blockconfig_1>>> (up_0, up_1, up_2, u_0, u_1, u_2, um_0, um_1, um_2, rho, strx, stry, strz, dcx, dcy, dcz, cox, coy, coz, L, M, N);

	hipMemcpy (h_up_0, up_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_up_1, up_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_up_2, up_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
}
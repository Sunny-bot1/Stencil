#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 4
#define by 4
#define bz 16

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void addsg (double * __restrict__ d_up_0, double * __restrict__ d_up_1, double * __restrict__ d_up_2, double * __restrict__ d_u_0, double * __restrict__ d_u_1, double * __restrict__ d_u_2, double * __restrict__ d_um_0, double * __restrict__ d_um_1, double * __restrict__ d_um_2, double * __restrict__ d_rho, double * __restrict__ strx, double * __restrict__ stry, double * __restrict__ strz, double * __restrict__ dcx, double * __restrict__ dcy, double * __restrict__ dcz, double * __restrict__ cox, double * __restrict__ coy, double * __restrict__ coz, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*(2*(int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(2*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(2*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*up_0)[MM][NN] = (double (*)[MM][NN]) d_up_0;
	double (*up_1)[MM][NN] = (double (*)[MM][NN]) d_up_1;
	double (*up_2)[MM][NN] = (double (*)[MM][NN]) d_up_2;
	double (*u_0)[MM][NN] = (double (*)[MM][NN]) d_u_0;
	double (*u_1)[MM][NN] = (double (*)[MM][NN]) d_u_1;
	double (*u_2)[MM][NN] = (double (*)[MM][NN]) d_u_2;
	double (*um_0)[MM][NN] = (double (*)[MM][NN]) d_um_0;
	double (*um_1)[MM][NN] = (double (*)[MM][NN]) d_um_1;
	double (*um_2)[MM][NN] = (double (*)[MM][NN]) d_um_2;
	double (*rho)[MM][NN] = (double (*)[MM][NN]) d_rho;

	//Array and variable declarations
	double birho;
	double beta = 0.625;

	#pragma unroll 2
	for (int r0_k2 = 0, k2 = k; r0_k2 < 2; r0_k2++, k2+=blockDim.z) {
		#pragma unroll 2
		for (int r0_j2 = 0, j2 = j; r0_j2 < 2; r0_j2++, j2+=blockDim.y) {
			#pragma unroll 2
			for (int r0_i2 = 0, i2 = i; r0_i2 < 2; r0_i2++, i2+=blockDim.x) {
				if (k2 >= max(2, k0) && k2 <= min(k0+2*blockDim.z-1, L-3) && j2 >= max(2, j0) && j2 <= min(j0+2*blockDim.y-1, M-3) && i2 >= max(2, i0) && i2 <= min(i0+2*blockDim.x-1, N-3)) {
					birho = beta / rho[k2][j2][i2];
					up_0[k2][j2][i2] -= birho * (strx[i2] * coy[j2] * coz[k2] * (rho[k2][j2][i2+1] * dcx[i2+1] * (u_0[k2][j2][i2+2] - 2 * u_0[k2][j2][i2+1] + u_0[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcx[i2] * (u_0[k2][j2][i2+1] - 2 * u_0[k2][j2][i2] + u_0[k2][j2][i2-1]) + rho[k2][j2][i2-1] * dcx[i2-1] * (u_0[k2][j2][i2] - 2 * u_0[k2][j2][i2-1] + u_0[k2][j2][i2-2]) - rho[k2][j2][i2+1] * dcx[i2+1] * (um_0[k2][j2][i2+2] - 2 * um_0[k2][j2][i2+1] + um_0[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcx[i2] * (um_0[k2][j2][i2+1] - 2 * um_0[k2][j2][i2] + um_0[k2][j2][i2-1]) - rho[k2][j2][i2-1] * dcx[i2-1] * (um_0[k2][j2][i2] - 2 * um_0[k2][j2][i2-1] + um_0[k2][j2][i2-2])) + stry[j2] * cox[i2] * coz[k2] * (rho[k2][j2+1][i2] * dcy[j2+1] * (u_0[k2][j2+2][i2] - 2 * u_0[k2][j2+1][i2] + u_0[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcy[j2] * (u_0[k2][j2+1][i2] - 2 * u_0[k2][j2][i2] + u_0[k2][j2-1][i2]) + rho[k2][j2-1][i2] * dcy[j2-1] * (u_0[k2][j2][i2] - 2 * u_0[k2][j2-1][i2] + u_0[k2][j2-2][i2]) - rho[k2][j2+1][i2] * dcy[j2+1] * (um_0[k2][j2+2][i2] - 2 * um_0[k2][j2+1][i2] + um_0[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcy[j2] * (um_0[k2][j2+1][i2] - 2 * um_0[k2][j2][i2] + um_0[k2][j2-1][i2]) - rho[k2][j2-1][i2] * dcy[j2-1] * (um_0[k2][j2][i2] - 2 * um_0[k2][j2-1][i2] + um_0[k2][j2-2][i2])) + strz[k2] * cox[i2] * coy[j2] * (rho[k2+1][j2][i2] * dcz[k2+1] * (u_0[k2+2][j2][i2] - 2 * u_0[k2+1][j2][i2] + u_0[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcz[k2] * (u_0[k2+1][j2][i2] - 2 * u_0[k2][j2][i2] + u_0[k2-1][j2][i2]) + rho[k2-1][j2][i2] * dcz[k2-1] * (u_0[k2][j2][i2] - 2 * u_0[k2-1][j2][i2] + u_0[k2-2][j2][i2]) - rho[k2+1][j2][i2] * dcz[k2+1] * (um_0[k2+2][j2][i2] - 2 * um_0[k2+1][j2][i2] + um_0[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcz[k2] * (um_0[k2+1][j2][i2] - 2 * um_0[k2][j2][i2] + um_0[k2-1][j2][i2]) - rho[k2-1][j2][i2] * dcz[k2-1] * (um_0[k2][j2][i2] - 2 * um_0[k2-1][j2][i2] + um_0[k2-2][j2][i2])));
					up_1[k2][j2][i2] -= birho * (strx[i2] * coy[j2] * coz[k2] * (rho[k2][j2][i2+1] * dcx[i2+1] * (u_1[k2][j2][i2+2] - 2 * u_1[k2][j2][i2+1] + u_1[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcx[i2] * (u_1[k2][j2][i2+1] - 2 * u_1[k2][j2][i2] + u_1[k2][j2][i2-1]) + rho[k2][j2][i2-1] * dcx[i2-1] * (u_1[k2][j2][i2] - 2 * u_1[k2][j2][i2-1] + u_1[k2][j2][i2-2]) - rho[k2][j2][i2+1] * dcx[i2+1] * (um_1[k2][j2][i2+2] - 2 * um_1[k2][j2][i2+1] + um_1[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcx[i2] * (um_1[k2][j2][i2+1] - 2 * um_1[k2][j2][i2] + um_1[k2][j2][i2-1]) - rho[k2][j2][i2-1] * dcx[i2-1] * (um_1[k2][j2][i2] - 2 * um_1[k2][j2][i2-1] + um_1[k2][j2][i2-2])) + stry[j2] * cox[i2] * coz[k2] * (rho[k2][j2+1][i2] * dcy[j2+1] * (u_1[k2][j2+2][i2] - 2 * u_1[k2][j2+1][i2] + u_1[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcy[j2] * (u_1[k2][j2+1][i2] - 2 * u_1[k2][j2][i2] + u_1[k2][j2-1][i2]) + rho[k2][j2-1][i2] * dcy[j2-1] * (u_1[k2][j2][i2] - 2 * u_1[k2][j2-1][i2] + u_1[k2][j2-2][i2]) - rho[k2][j2+1][i2] * dcy[j2+1] * (um_1[k2][j2+2][i2] - 2 * um_1[k2][j2+1][i2] + um_1[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcy[j2] * (um_1[k2][j2+1][i2] - 2 * um_1[k2][j2][i2] + um_1[k2][j2-1][i2]) - rho[k2][j2-1][i2] * dcy[j2-1] * (um_1[k2][j2][i2] - 2 * um_1[k2][j2-1][i2] + um_1[k2][j2-2][i2])) + strz[k2] * cox[i2] * coy[j2] * (rho[k2+1][j2][i2] * dcz[k2+1] * (u_1[k2+2][j2][i2] - 2 * u_1[k2+1][j2][i2] + u_1[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcz[k2] * (u_1[k2+1][j2][i2] - 2 * u_1[k2][j2][i2] + u_1[k2-1][j2][i2]) + rho[k2-1][j2][i2] * dcz[k2-1] * (u_1[k2][j2][i2] - 2 * u_1[k2-1][j2][i2] + u_1[k2-2][j2][i2]) - rho[k2+1][j2][i2] * dcz[k2+1] * (um_1[k2+2][j2][i2] - 2 * um_1[k2+1][j2][i2] + um_1[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcz[k2] * (um_1[k2+1][j2][i2] - 2 * um_1[k2][j2][i2] + um_1[k2-1][j2][i2]) - rho[k2-1][j2][i2] * dcz[k2-1] * (um_1[k2][j2][i2] - 2 * um_1[k2-1][j2][i2] + um_1[k2-2][j2][i2])));
					up_2[k2][j2][i2] -= birho * (strx[i2] * coy[j2] * coz[k2] * (rho[k2][j2][i2+1] * dcx[i2+1] * (u_2[k2][j2][i2+2] - 2 * u_2[k2][j2][i2+1] + u_2[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcx[i2] * (u_2[k2][j2][i2+1] - 2 * u_2[k2][j2][i2] + u_2[k2][j2][i2-1]) + rho[k2][j2][i2-1] * dcx[i2-1] * (u_2[k2][j2][i2] - 2 * u_2[k2][j2][i2-1] + u_2[k2][j2][i2-2]) - rho[k2][j2][i2+1] * dcx[i2+1] * (um_2[k2][j2][i2+2] - 2 * um_2[k2][j2][i2+1] + um_2[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcx[i2] * (um_2[k2][j2][i2+1] - 2 * um_2[k2][j2][i2] + um_2[k2][j2][i2-1]) - rho[k2][j2][i2-1] * dcx[i2-1] * (um_2[k2][j2][i2] - 2 * um_2[k2][j2][i2-1] + um_2[k2][j2][i2-2])) + stry[j2] * cox[i2] * coz[k2] * (rho[k2][j2+1][i2] * dcy[j2+1] * (u_2[k2][j2+2][i2] - 2 * u_2[k2][j2+1][i2] + u_2[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcy[j2] * (u_2[k2][j2+1][i2] - 2 * u_2[k2][j2][i2] + u_2[k2][j2-1][i2]) + rho[k2][j2-1][i2] * dcy[j2-1] * (u_2[k2][j2][i2] - 2 * u_2[k2][j2-1][i2] + u_2[k2][j2-2][i2]) - rho[k2][j2+1][i2] * dcy[j2+1] * (um_2[k2][j2+2][i2] - 2 * um_2[k2][j2+1][i2] + um_2[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcy[j2] * (um_2[k2][j2+1][i2] - 2 * um_2[k2][j2][i2] + um_2[k2][j2-1][i2]) - rho[k2][j2-1][i2] * dcy[j2-1] * (um_2[k2][j2][i2] - 2 * um_2[k2][j2-1][i2] + um_2[k2][j2-2][i2])) + strz[k2] * cox[i2] * coy[j2] * (rho[k2+1][j2][i2] * dcz[k2+1] * (u_2[k2+2][j2][i2] - 2 * u_2[k2+1][j2][i2] + u_2[k2][j2][i2]) - 2 * rho[k2][j2][i2] * dcz[k2] * (u_2[k2+1][j2][i2] - 2 * u_2[k2][j2][i2] + u_2[k2-1][j2][i2]) + rho[k2-1][j2][i2] * dcz[k2-1] * (u_2[k2][j2][i2] - 2 * u_2[k2-1][j2][i2] + u_2[k2-2][j2][i2]) - rho[k2+1][j2][i2] * dcz[k2+1] * (um_2[k2+2][j2][i2] - 2 * um_2[k2+1][j2][i2] + um_2[k2][j2][i2]) + 2 * rho[k2][j2][i2] * dcz[k2] * (um_2[k2+1][j2][i2] - 2 * um_2[k2][j2][i2] + um_2[k2-1][j2][i2]) - rho[k2-1][j2][i2] * dcz[k2-1] * (um_2[k2][j2][i2] - 2 * um_2[k2-1][j2][i2] + um_2[k2-2][j2][i2])));
				}
			}
		}
	}
}


extern "C" void host_code (double *h_up_0, double *h_up_1, double *h_up_2, double *h_u_0, double *h_u_1, double *h_u_2, double *h_um_0, double *h_um_1, double *h_um_2, double *h_rho, double *h_strx, double *h_stry, double *h_strz, double *h_dcx, double *h_dcy, double *h_dcz, double *h_cox, double *h_coy, double *h_coz, int L, int M, int N) {
	double *up_0;
	hipMalloc (&up_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for up_0\n");
	hipMemcpy (up_0, h_up_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *up_1;
	hipMalloc (&up_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for up_1\n");
	hipMemcpy (up_1, h_up_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *up_2;
	hipMalloc (&up_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for up_2\n");
	hipMemcpy (up_2, h_up_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *u_0;
	hipMalloc (&u_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for u_0\n");
	hipMemcpy (u_0, h_u_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *u_1;
	hipMalloc (&u_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for u_1\n");
	hipMemcpy (u_1, h_u_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *u_2;
	hipMalloc (&u_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for u_2\n");
	hipMemcpy (u_2, h_u_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *um_0;
	hipMalloc (&um_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for um_0\n");
	hipMemcpy (um_0, h_um_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *um_1;
	hipMalloc (&um_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for um_1\n");
	hipMemcpy (um_1, h_um_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *um_2;
	hipMalloc (&um_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for um_2\n");
	hipMemcpy (um_2, h_um_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *rho;
	hipMalloc (&rho, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for rho\n");
	hipMemcpy (rho, h_rho, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *strx;
	hipMalloc (&strx, sizeof(double )*(N - 0));
	check_error ("Failed to allocate device memory for strx\n");
	hipMemcpy (strx, h_strx, sizeof(double )*(N - 0), hipMemcpyHostToDevice);
	double *stry;
	hipMalloc (&stry, sizeof(double )*(M - 0));
	check_error ("Failed to allocate device memory for stry\n");
	hipMemcpy (stry, h_stry, sizeof(double )*(M - 0), hipMemcpyHostToDevice);
	double *strz;
	hipMalloc (&strz, sizeof(double )*(L - 0));
	check_error ("Failed to allocate device memory for strz\n");
	hipMemcpy (strz, h_strz, sizeof(double )*(L - 0), hipMemcpyHostToDevice);
	double *dcx;
	hipMalloc (&dcx, sizeof(double )*(N - 0));
	check_error ("Failed to allocate device memory for dcx\n");
	hipMemcpy (dcx, h_dcx, sizeof(double )*(N - 0), hipMemcpyHostToDevice);
	double *dcy;
	hipMalloc (&dcy, sizeof(double )*(M - 0));
	check_error ("Failed to allocate device memory for dcy\n");
	hipMemcpy (dcy, h_dcy, sizeof(double )*(M - 0), hipMemcpyHostToDevice);
	double *dcz;
	hipMalloc (&dcz, sizeof(double )*(L - 0));
	check_error ("Failed to allocate device memory for dcz\n");
	hipMemcpy (dcz, h_dcz, sizeof(double )*(L - 0), hipMemcpyHostToDevice);
	double *cox;
	hipMalloc (&cox, sizeof(double )*(N - 0));
	check_error ("Failed to allocate device memory for cox\n");
	hipMemcpy (cox, h_cox, sizeof(double )*(N - 0), hipMemcpyHostToDevice);
	double *coy;
	hipMalloc (&coy, sizeof(double )*(M - 0));
	check_error ("Failed to allocate device memory for coy\n");
	hipMemcpy (coy, h_coy, sizeof(double )*(M - 0), hipMemcpyHostToDevice);
	double *coz;
	hipMalloc (&coz, sizeof(double )*(L - 0));
	check_error ("Failed to allocate device memory for coz\n");
	hipMemcpy (coz, h_coz, sizeof(double )*(L - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, 2*blockconfig_1.x), ceil (M - 1 - 0 + 1, 2*blockconfig_1.y), ceil (L - 1 - 0 + 1, 2*blockconfig_1.z));

	addsg <<<gridconfig_1, blockconfig_1>>> (up_0, up_1, up_2, u_0, u_1, u_2, um_0, um_1, um_2, rho, strx, stry, strz, dcx, dcy, dcz, cox, coy, coz, L, M, N);

	hipMemcpy (h_up_0, up_0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_up_1, up_1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_up_2, up_2, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
}
#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 8
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave1 (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*(2*(int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations
	#pragma unroll 2
	for (int r0_i2 = 0, i2 = i; r0_i2 < 2; r0_i2++, i2+=blockDim.x) {
		if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i2 >= i0 && i2 <= min(i0+2*blockDim.x-1, N-2)) {
			ppgu[k][j][i2] = (ppuv[k][j][i2+1] - ppuv[k][j][i2]) + (ppgc[k][j][i2+1] + ppgc[k][j][i2]) * 0.5 * ((hhl[k+1][j][i2] + hhl[k][j][i]) - (hhl[k+1][j][i2+1] + hhl[k][j][i2+1])) / ((hhl[k+1][j][i2] - hhl[k][j][i2]) + (hhl[k+1][j][i2+1] + hhl[k][j][i2+1]));
		}
	}
}

__global__ void fastwave2 (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*(2*(int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations
	#pragma unroll 2
	for (int r0_i2 = 0, i2 = i; r0_i2 < 2; r0_i2++, i2+=blockDim.x) {
		if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-2) && i2 >= i0 && i2 <= min(i0+2*blockDim.x-1, N-2)) {
			ppgu[k][j][i2] = (ppuv[k][j+1][i2] - ppuv[k][j][i2]) + (ppgc[k][j+1][i2] + ppgc[k][j][i2]) * 0.5 * ((hhl[k+1][j][i2] + hhl[k][j][i2]) - (hhl[k+1][j+1][i2] + hhl[k][j+1][i2])) / ((hhl[k+1][j][i2] - hhl[k][j][i2]) + (hhl[k+1][j+1][i2] + hhl[k][j+1][i2]));
		}
	}
}


extern "C" void host_code (double *h_ppgu0, double *h_ppgu1,double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {

	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipStream_t s1, s2;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	
	double *ppgu0;
	hipMalloc (&ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu0\n");
	//cudaMemcpy (ppgu0, h_ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), cudaMemcpyHostToDevice);
	double *ppgu1;
	hipMalloc (&ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu1\n");
	//cudaMemcpy (ppgu1, h_ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), cudaMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpyAsync (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpyAsync (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpyAsync (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, 2*blockconfig_1.x), ceil (M - 1 - 0 + 1, blockconfig_1.y), ceil (L - 1 - 0 + 1, blockconfig_1.z));
	hipEventRecord(start);
	fastwave1 <<<gridconfig_1, blockconfig_1>>> (ppgu0, ppuv, ppgc, hhl, L, M, N);
	hipError_t err0 = hipGetLastError();
	if (err0 != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(err0));
		// Possibly: exit(-1) if program cannot continue....
	} 
	fastwave2 <<<gridconfig_1, blockconfig_1>>> (ppgu1, ppuv, ppgc, hhl, L, M, N);
	hipError_t err1 = hipGetLastError();
	if (err1 != hipSuccess) {
		printf("CUDA Error: %s\n", hipGetErrorString(err1));
		// Possibly: exit(-1) if program cannot continue....
	} 
	hipEventRecord(stop);
	hipMemcpyAsync (h_ppgu0, ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpyAsync (h_ppgu1, ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
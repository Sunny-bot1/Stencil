#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 64
#define by 4
#define bz 4

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(8*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(2*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 2
	for (int r0_k2 = 0, k2 = k; r0_k2 < 2; r0_k2++, k2+=blockDim.z) {
		#pragma unroll 8
		for (int r0_j8 = 0, j8 = j; r0_j8 < 8; r0_j8++, j8+=blockDim.y) {
			if (k2 >= k0 && k2 <= min(k0+2*blockDim.z-1, L-2) && j8 >= j0 && j8 <= min(j0+8*blockDim.y-1, M-2) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
				ppgu[k2][j8][i] = (ppuv[k2][j8+1][i] - ppuv[k2][j8][i]) + (ppgc[k2][j8+1][i] + ppgc[k2][j8][i]) * 0.5 * ((hhl[k2+1][j8][i] + hhl[k2][j8][i]) - (hhl[k2+1][j8+1][i] + hhl[k2][j8+1][i])) / ((hhl[k2+1][j8][i] - hhl[k2][j8][i]) + (hhl[k2+1][j8+1][i] + hhl[k2][j8+1][i]));
			}
		}
	}
}


extern "C" void host_code (double *h_ppgu, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	double *ppgu;
	hipMalloc (&ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu\n");
	hipMemcpy (ppgu, h_ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, 8*blockconfig_1.y), ceil (L - 1 - 0 + 1, 2*blockconfig_1.z));

	hipEvent_t start, stop; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    fastwave <<<gridconfig_1, blockconfig_1>>> (ppgu, ppuv, ppgc, hhl, L, M, N);
    printf("%d,%d,%d,%d,", gridconfig_1.x, gridconfig_1.y, gridconfig_1.z, gridconfig_1.x*gridconfig_1.y*gridconfig_1.z);
    hipEventRecord(stop);
    hipMemcpy (h_ppgu, ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
    float time;
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("%f\n", time);

}
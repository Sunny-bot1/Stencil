#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 128
#define MM 128
#define NN 128

#define bx 16
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fusion (double * __restrict__ d_ppgu0, double * __restrict__ d_ppgu1, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu0)[MM][NN] = (double (*)[MM][NN]) d_ppgu0;
	double (*ppgu1)[MM][NN] = (double (*)[MM][NN]) d_ppgu1;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-2)) {
		ppgu0[k][j][i] = (ppuv[k][j][i+1] - ppuv[k][j][i]) + (ppgc[k][j][i+1] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j][i+1] + hhl[k][j][i+1])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j][i+1] + hhl[k][j][i+1]));
	}
	if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-2) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
		ppgu1[k][j][i] = (ppuv[k][j][i+1] - ppuv[k][j][i]) + (ppgc[k][j][i+1] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j][i+1] + hhl[k][j][i+1])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j][i+1] + hhl[k][j][i+1]));
	}
}


extern "C" void host_code (double *h_ppgu0, double *h_ppgu1, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	double *ppgu0;
	hipMalloc (&ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu0\n");
	hipMemcpy (ppgu0, h_ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgu1;
	hipMalloc (&ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu1\n");
	hipMemcpy (ppgu1, h_ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, blockconfig_1.y), ceil (L - 1 - 0 + 1, blockconfig_1.z));
	hipEventRecord(start);
	fusion <<<gridconfig_1, blockconfig_1>>> (ppgu0, ppgu1, ppuv, ppgc, hhl, L, M, N);
	hipEventRecord(stop);
	hipMemcpy (h_ppgu0, ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_ppgu1, ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
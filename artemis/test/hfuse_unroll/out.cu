#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 128
#define MM 128
#define NN 128

#define bx 16
#define by 8
#define bz 8

#define bx1 10

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fusion (double * __restrict__ d_ppgu0, double * __restrict__ d_ppgu1, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
	int tid_x1 = threadIdx.x;
	int tid_x2 = threadIdx.x - bx1;
	int bdim_x1 = bx1;
	int bdim_x2 = bx - bx1;
	//Determining the block's indices
	// int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	// int i = i0 + (int)(threadIdx.x);
	// int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	// int j = j0 + (int)(threadIdx.y);
	// int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	// int k = k0 + (int)(threadIdx.z);

	double (*ppgu0)[MM][NN] = (double (*)[MM][NN]) d_ppgu0;
	double (*ppgu1)[MM][NN] = (double (*)[MM][NN]) d_ppgu1;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations
	if(threadIdx.x < bx1){
		int i0 = (int)(blockIdx.x)*bdim_x1;
		int i = i0 + tid_x1;
		int j0 = (int)(blockIdx.y)*((int)blockDim.y);
		int j = j0 + (int)(threadIdx.y);
		int k0 = (int)(blockIdx.z)*((int)blockDim.z);
		int k = k0 + (int)(threadIdx.z);
		if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+bdim_x1-1, N-2)) {
			ppgu0[k][j][i] = (ppuv[k][j][i+1] - ppuv[k][j][i]) + (ppgc[k][j][i+1] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j][i+1] + hhl[k][j][i+1])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j][i+1] + hhl[k][j][i+1]));
		}
		return;
	}
	else{
		int i0 = (int)(blockIdx.x)*2*bdim_x2;
		int i = i0 + tid_x2;
		int j0 = (int)(blockIdx.y)*((int)blockDim.y);
		int j = j0 + (int)(threadIdx.y);
		int k0 = (int)(blockIdx.z)*((int)blockDim.z);
		int k = k0 + (int)(threadIdx.z);
		#pragma unroll 2
		for (int r0_i2 = 0, i2 = i; r0_i2 < 2; r0_i2++, i2+=bdim_x2) {
			if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-2) && i2 >= i0 && i2 <= min(i0+2*bdim_x2-1, N-1)) {
				ppgu1[k][j][i2] = (ppuv[k][j+1][i2] - ppuv[k][j][i2]) + (ppgc[k][j+1][i2] + ppgc[k][j][i2]) * 0.5 * ((hhl[k+1][j][i2] + hhl[k][j][i2]) - (hhl[k+1][j+1][i2] + hhl[k][j+1][i2])) / ((hhl[k+1][j][i2] - hhl[k][j][i2]) + (hhl[k+1][j+1][i2] + hhl[k][j+1][i2]));
			}
		}
	}
}


extern "C" void host_code (double *h_ppgu0, double *h_ppgu1, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	double *ppgu0;
	hipMalloc (&ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu0\n");
	hipMemcpy (ppgu0, h_ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgu1;
	hipMalloc (&ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu1\n");
	hipMemcpy (ppgu1, h_ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz); 
	dim3 gridconfig_1 (ceil (N, bx/2), ceil (M, blockconfig_1.y), ceil (L, blockconfig_1.z));

	hipEventRecord(start);
	fusion <<<gridconfig_1, blockconfig_1>>> (ppgu0, ppgu1, ppuv, ppgc, hhl, L, M, N);
	hipEventRecord(stop);
	hipMemcpy (h_ppgu0, ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_ppgu1, ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	printf("BlockDim1 (%d, %d, %d)\n", blockconfig_1.x, blockconfig_1.y, blockconfig_1.y);
	printf("GridDim (%d, %d, %d)\n", gridconfig_1.x, gridconfig_1.y, gridconfig_1.y);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
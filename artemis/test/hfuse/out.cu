#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 256
#define MM 256
#define NN 256

#define bx 16
#define by 8
#define bz 8

#define bx1 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fusion (double * __restrict__ d_ppgu0, double * __restrict__ d_ppgu1, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
	int threadIdx_x, threadIdx_y, threadIdx_z;
	int blockDim_x, blockDim_y, blockDim_z;
	if(tid < 512){
		blockDim_x = 8;
		blockDim_y = 8;
		blockDim_z = 8;
		threadIdx_x = tid % blockDim_x;
		threadIdx_y = tid / blockDim_x % blockDim_y;
		threadIdx_z = tid / blockDim_x / blockDim_y % blockDim_z;
	}
	else{
		blockDim_x = 8;
		blockDim_y = 8;
		blockDim_z = 8;
		threadIdx_x = (tid - 512) % blockDim_x;
		threadIdx_y = (tid - 512) / blockDim_x % blockDim_y;
		threadIdx_z = (tid - 512) / blockDim_x / blockDim_y % blockDim_z;
	}
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim_x);
	int i = i0 + (int)(threadIdx_x);
	int j0 = (int)(blockIdx.y)*((int)blockDim_y);
	int j = j0 + (int)(threadIdx_y);
	int k0 = (int)(blockIdx.z)*((int)blockDim_z);
	int k = k0 + (int)(threadIdx_z);

	// double (*ppgu0)[MM][NN] = (double (*)[MM][NN]) d_ppgu0;
	// double (*ppgu1)[MM][NN] = (double (*)[MM][NN]) d_ppgu1;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations
	if(tid < 512){
		double (*ppgu0)[MM][NN] = (double (*)[MM][NN]) d_ppgu0;
		if (k >= k0 && k <= min(k0+blockDim_z-1, L-2) && j >= j0 && j <= min(j0+blockDim_y-1, M-1) && i >= i0 && i <= min(i0+blockDim_x-1, N-2)) {
			ppgu0[k][j][i] = (ppuv[k][j][i+1] - ppuv[k][j][i]) + (ppgc[k][j][i+1] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j][i+1] + hhl[k][j][i+1])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j][i+1] + hhl[k][j][i+1]));
		}
	}
	else{
		double (*ppgu1)[MM][NN] = (double (*)[MM][NN]) d_ppgu1;
		if (k >= k0 && k <= min(k0+blockDim_z-1, L-2) && j >= j0 && j <= min(j0+blockDim_y-1, M-2) && i >= i0 && i <= min(i0+blockDim_x-1, N-1)) {
			ppgu1[k][j][i] = (ppuv[k][j+1][i] - ppuv[k][j][i]) + (ppgc[k][j+1][i] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j+1][i] + hhl[k][j+1][i])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j+1][i] + hhl[k][j+1][i]));
		}
	}
}


extern "C" void host_code (double *h_ppgu0, double *h_ppgu1, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	double *ppgu0;
	hipMalloc (&ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu0\n");
	hipMemcpy (ppgu0, h_ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgu1;
	hipMalloc (&ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu1\n");
	hipMemcpy (ppgu1, h_ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz); 
	dim3 gridconfig_1 (ceil (N, min(bx1, bx-bx1)), ceil (M, blockconfig_1.y), ceil (L, blockconfig_1.z));

	hipEventRecord(start);
	fusion <<<gridconfig_1, blockconfig_1>>> (ppgu0, ppgu1, ppuv, ppgc, hhl, L, M, N);
	hipEventRecord(stop);
	hipMemcpy (h_ppgu0, ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_ppgu1, ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	printf("BlockDim1 (%d, %d, %d)\n", blockconfig_1.x, blockconfig_1.y, blockconfig_1.y);
	printf("GridDim (%d, %d, %d)\n", gridconfig_1.x, gridconfig_1.y, gridconfig_1.y);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
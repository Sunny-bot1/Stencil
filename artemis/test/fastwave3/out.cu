#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 32
#define by 4
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(16*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(2*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 2
	for (int r0_k2 = 0, k2 = k; r0_k2 < 2; r0_k2++, k2+=blockDim.z) {
		#pragma unroll 16
		for (int r0_j16 = 0, j16 = j; r0_j16 < 16; r0_j16++, j16+=blockDim.y) {
			if (k2 >= k0 && k2 <= min(k0+2*blockDim.z-1, L-2) && j16 >= j0 && j16 <= min(j0+16*blockDim.y-1, M-2) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
				ppgu[k2][j16][i] = (ppuv[k2][j16+1][i] - ppuv[k2][j16][i]) + (ppgc[k2][j16+1][i] + ppgc[k2][j16][i]) * 0.5 * ((hhl[k2+1][j16][i] + hhl[k2][j16][i]) - (hhl[k2+1][j16+1][i] + hhl[k2][j16+1][i])) / ((hhl[k2+1][j16][i] - hhl[k2][j16][i]) + (hhl[k2+1][j16+1][i] + hhl[k2][j16+1][i]));
			}
		}
	}
}


extern "C" void host_code (double *h_ppgu, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	double *ppgu;
	hipMalloc (&ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu\n");
	hipMemcpy (ppgu, h_ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, 16*blockconfig_1.y), ceil (L - 1 - 0 + 1, 2*blockconfig_1.z));

	fastwave <<<gridconfig_1, blockconfig_1>>> (ppgu, ppuv, ppgc, hhl, L, M, N);

	hipMemcpy (h_ppgu, ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
}
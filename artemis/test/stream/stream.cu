#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 32
#define by 4
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave1 (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(16*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(2*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 2
	for (int r0_k2 = 0, k2 = k; r0_k2 < 2; r0_k2++, k2+=blockDim.z) {
		#pragma unroll 16
		for (int r0_j16 = 0, j16 = j; r0_j16 < 16; r0_j16++, j16+=blockDim.y) {
			if (k2 >= k0 && k2 <= min(k0+2*blockDim.z-1, L-2) && j16 >= j0 && j16 <= min(j0+16*blockDim.y-1, M-2) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
				ppgu[k2][j16][i] = (ppuv[k2][j16+1][i] - ppuv[k2][j16][i]) + (ppgc[k2][j16+1][i] + ppgc[k2][j16][i]) * 0.5 * ((hhl[k2+1][j16][i] + hhl[k2][j16][i]) - (hhl[k2+1][j16+1][i] + hhl[k2][j16+1][i])) / ((hhl[k2+1][j16][i] - hhl[k2][j16][i]) + (hhl[k2+1][j16+1][i] + hhl[k2][j16+1][i]));
			}
		}
	}
}

__global__ void fastwave2 (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(16*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(2*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 2
	for (int r0_k2 = 0, k2 = k; r0_k2 < 2; r0_k2++, k2+=blockDim.z) {
		#pragma unroll 16
		for (int r0_j16 = 0, j16 = j; r0_j16 < 16; r0_j16++, j16+=blockDim.y) {
			if (k2 >= k0 && k2 <= min(k0+2*blockDim.z-1, L-2) && j16 >= j0 && j16 <= min(j0+16*blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-2)) {
				ppgu[k2][j16][i] = (ppuv[k2][j16][i+1] - ppuv[k2][j16][i]) + (ppgc[k2][j16][i+1] + ppgc[k2][j16][i]) * 0.5 * ((hhl[k2+1][j16][i] + hhl[k2][j16][i]) - (hhl[k2+1][j16][i+1] + hhl[k2][j16][i+1])) / ((hhl[k2+1][j16][i] - hhl[k2][j16][i]) + (hhl[k2+1][j16][i+1] + hhl[k2][j16][i+1]));
			}
		}
	}
}


extern "C" void host_code (double *h_ppgu0, double *h_ppgu1,double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipStream_t s1, s2;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	
	double *ppgu0;
	hipMalloc (&ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu0\n");
	//cudaMemcpy (ppgu0, h_ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), cudaMemcpyHostToDevice);
	double *ppgu1;
	hipMalloc (&ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu1\n");
	//cudaMemcpy (ppgu1, h_ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), cudaMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpyAsync (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpyAsync (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpyAsync (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, 16*blockconfig_1.y), ceil (L - 1 - 0 + 1, 2*blockconfig_1.z));
	hipEventRecord(start);
	fastwave1 <<<gridconfig_1, blockconfig_1, 0, s1>>> (ppgu0, ppuv, ppgc, hhl, L, M, N);
	fastwave2 <<<gridconfig_1, blockconfig_1, 0, s2>>> (ppgu1, ppuv, ppgc, hhl, L, M, N);
	hipEventRecord(stop);
	hipMemcpyAsync (h_ppgu0, ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost, s1);
	hipMemcpyAsync (h_ppgu1, ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost, s2);
	
	hipStreamDestroy(s1);
	hipStreamDestroy(s2);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
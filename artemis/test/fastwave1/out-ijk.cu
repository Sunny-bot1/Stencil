#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 16
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*(2*(int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(2*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(2*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 2
	for (int r0_k2 = 0, k2 = k; r0_k2 < 2; r0_k2++, k2+=blockDim.z) {
		#pragma unroll 2
		for (int r0_j2 = 0, j2 = j; r0_j2 < 2; r0_j2++, j2+=blockDim.y) {
			#pragma unroll 2
			for (int r0_i2 = 0, i2 = i; r0_i2 < 2; r0_i2++, i2+=blockDim.x) {
				if (k2 >= k0 && k2 <= min(k0+2*blockDim.z-1, L-2) && j2 >= j0 && j2 <= min(j0+2*blockDim.y-1, M-1) && i2 >= i0 && i2 <= min(i0+2*blockDim.x-1, N-2)) {
					ppgu[k2][j2][i2] = (ppuv[k2][j2][i2+1] - ppuv[k2][j2][i2]) + (ppgc[k2][j2][i2+1] + ppgc[k2][j2][i2]) * 0.5 * ((hhl[k2+1][j2][i2] + hhl[k2][j2][i2]) - (hhl[k2+1][j2][i2+1] + hhl[k2][j2][i2+1])) / ((hhl[k2+1][j2][i2] - hhl[k2][j2][i2]) + (hhl[k2+1][j2][i2+1] + hhl[k2][j2][i2+1]));
				}
			}
		}
	}
}


extern "C" void host_code (double *h_ppgu, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	double *ppgu;
	hipMalloc (&ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu\n");
	hipMemcpy (ppgu, h_ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, 2*blockconfig_1.x), ceil (M - 1 - 0 + 1, 2*blockconfig_1.y), ceil (L - 1 - 0 + 1, 2*blockconfig_1.z));

	fastwave <<<gridconfig_1, blockconfig_1>>> (ppgu, ppuv, ppgc, hhl, L, M, N);

	hipMemcpy (h_ppgu, ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
}
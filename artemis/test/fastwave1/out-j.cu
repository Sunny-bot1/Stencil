#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 8
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(2*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 2
	for (int r0_j2 = 0, j2 = j; r0_j2 < 2; r0_j2++, j2+=blockDim.y) {
		if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j2 >= j0 && j2 <= min(j0+2*blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-2)) {
			ppgu[k][j2][i] = (ppuv[k][j2][i+1] - ppuv[k][j2][i]) + (ppgc[k][j2][i+1] + ppgc[k][j2][i]) * 0.5 * ((hhl[k+1][j2][i] + hhl[k][j2][i]) - (hhl[k+1][j2][i+1] + hhl[k][j2][i+1])) / ((hhl[k+1][j2][i] - hhl[k][j2][i]) + (hhl[k+1][j2][i+1] + hhl[k][j2][i+1]));
		}
	}
}


extern "C" void host_code (double *h_ppgu, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	double *ppgu;
	hipMalloc (&ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu\n");
	hipMemcpy (ppgu, h_ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, 2*blockconfig_1.y), ceil (L - 1 - 0 + 1, blockconfig_1.z));

	fastwave <<<gridconfig_1, blockconfig_1>>> (ppgu, ppuv, ppgc, hhl, L, M, N);

	hipMemcpy (h_ppgu, ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
}
#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 256
#define MM 256
#define NN 256

#define bx 16
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgc, double * __restrict__ d_ppgk, double * __restrict__ d_wgtfac, double * __restrict__ d_ppuv, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z-1);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*ppgk)[MM][NN] = (double (*)[MM][NN]) d_ppgk;
	double (*wgtfac)[MM][NN] = (double (*)[MM][NN]) d_wgtfac;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;

	//Array and variable declarations

	if (k >= max(1, k0) && k <= min(k0+blockDim.z-1, L-1) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
		ppgk[k][j][i] = wgtfac[k][j][i] * ppuv[k][j][i] + (1.0 - wgtfac[k][j][i]) * ppuv[k-1][j][i];
	}
	__syncthreads ();
	if (k >= max(1, k0) && k <= min(k0+blockDim.z-2, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
		ppgc[k][j][i] = ppgk[k+1][j][i] - ppgk[k][j][i];
	}
}


extern "C" void host_code (double *h_ppgc, double *h_ppgk, double *h_wgtfac, double *h_ppuv, int L, int M, int N) {
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgk;
	hipMalloc (&ppgk, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgk\n");
	hipMemcpy (ppgk, h_ppgk, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *wgtfac;
	hipMalloc (&wgtfac, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for wgtfac\n");
	hipMemcpy (wgtfac, h_wgtfac, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, blockconfig_1.y), ceil (L - 1 - 0 + 1, blockconfig_1.z-1));
	hipEventRecord(start);
	fastwave <<<gridconfig_1, blockconfig_1>>> (ppgc, ppgk, wgtfac, ppuv, L, M, N);
	hipEventRecord(stop);
	hipMemcpy (h_ppgc, ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_ppgk, ppgk, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
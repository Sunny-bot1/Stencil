#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 64
#define MM 64
#define NN 64

#define bx 16
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-2) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
		ppgu[k][j][i] = (ppuv[k][j+1][i] - ppuv[k][j][i])
						 + (ppgc[k][j+1][i] + ppgc[k][j][i]) 
						 * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j+1][i] + hhl[k][j+1][i]))
						  / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j+1][i] + hhl[k][j+1][i]));
	}
}


extern "C" void host_code (double *h_ppgu, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);

	double *ppgu;
	hipMalloc (&ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu\n");
	hipMemcpy (ppgu, h_ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, blockconfig_1.y), ceil (L - 1 - 0 + 1, blockconfig_1.z));
	hipEventRecord(start);
	fastwave <<<gridconfig_1, blockconfig_1>>> (ppgu, ppuv, ppgc, hhl, L, M, N);
	hipEventRecord(stop);
	hipMemcpy (h_ppgu, ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	printf("gridDim{%d, %d, %d}\n", gridconfig_1.x, gridconfig_1.y, gridconfig_1.z);
}
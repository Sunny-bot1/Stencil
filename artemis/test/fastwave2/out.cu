#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 128
#define MM 128
#define NN 128

#define bx 16
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*(4*(int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*(4*(int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*(4*(int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	#pragma unroll 4
	for (int r0_k4 = 0, k4 = k; r0_k4 < 4; r0_k4++, k4+=blockDim.z) {
		#pragma unroll 4
		for (int r0_j4 = 0, j4 = j; r0_j4 < 4; r0_j4++, j4+=blockDim.y) {
			#pragma unroll 4
			for (int r0_i4 = 0, i4 = i; r0_i4 < 4; r0_i4++, i4+=blockDim.x) {
				if (k4 >= k0 && k4 <= min(k0+4*blockDim.z-1, L-2) && j4 >= j0 && j4 <= min(j0+4*blockDim.y-1, M-2) && i4 >= i0 && i4 <= min(i0+4*blockDim.x-1, N-1)) {
					ppgu[k4][j4][i4] = (ppuv[k4][j4+1][i4] - ppuv[k4][j4][i4]) + (ppgc[k4][j4+1][i4] + ppgc[k4][j4][i4]) * 0.5 * ((hhl[k4+1][j4][i4] + hhl[k4][j4][i4]) - (hhl[k4+1][j4+1][i4] + hhl[k4][j4+1][i4])) / ((hhl[k4+1][j4][i4] - hhl[k4][j4][i4]) + (hhl[k4+1][j4+1][i4] + hhl[k4][j4+1][i4]));
				}
			}
		}
	}
}


extern "C" void host_code (double *h_ppgu, double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	double *ppgu;
	hipMalloc (&ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu\n");
	hipMemcpy (ppgu, h_ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpy (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, 4*blockconfig_1.x), ceil (M - 1 - 0 + 1, 4*blockconfig_1.y), ceil (L - 1 - 0 + 1, 4*blockconfig_1.z));

	fastwave <<<gridconfig_1, blockconfig_1>>> (ppgu, ppuv, ppgc, hhl, L, M, N);

	hipMemcpy (h_ppgu, ppgu, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
}
#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 256
#define MM 256
#define NN 256

#define bx 16
#define by 8
#define bz 8

#define bx1 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fusion (double * __restrict__ d_ppgc, double * __restrict__ d_ppgk, double * __restrict__ d_wgtfac, double * __restrict__ d_ppuv, int L, int M, int N) {
	int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
	int threadIdx_x, threadIdx_y, threadIdx_z;
	int blockDim_x, blockDim_y, blockDim_z;
	if(tid < 512){
		blockDim_x = 8;
		blockDim_y = 8;
		blockDim_z = 8;
		threadIdx_x = tid % blockDim_x;
		threadIdx_y = tid / blockDim_x % blockDim_y;
		threadIdx_z = tid / blockDim_x / blockDim_y % blockDim_z;
	}
	else{
		blockDim_x = 8;
		blockDim_y = 8;
		blockDim_z = 8;
		threadIdx_x = (tid - 512) % blockDim_x;
		threadIdx_y = (tid - 512) / blockDim_x % blockDim_y;
		threadIdx_z = (tid - 512) / blockDim_x / blockDim_y % blockDim_z;
	}
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim_x);
	int i = i0 + (int)(threadIdx_x);
	int j0 = (int)(blockIdx.y)*((int)blockDim_y);
	int j = j0 + (int)(threadIdx_y);
	int k0 = (int)(blockIdx.z)*((int)blockDim_z);
	int k = k0 + (int)(threadIdx_z);

	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*ppgk)[MM][NN] = (double (*)[MM][NN]) d_ppgk;
	double (*wgtfac)[MM][NN] = (double (*)[MM][NN]) d_wgtfac;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;

	//Array and variable declarations
	if(tid < 512){
		if (k >= max(1, k0) && k <= min(k0+blockDim.z-1, L-1) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
			ppgk[k][j][i] = wgtfac[k][j][i] * ppuv[k][j][i] + (1.0 - wgtfac[k][j][i]) * ppuv[k-1][j][i];
		}
		asm("bar.sync 1, 512;");
	}
	else{
		if (k >= max(1, k0) && k <= min(k0+blockDim.z-2, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
			ppgc[k][j][i] = ppgk[k+1][j][i] - ppgk[k][j][i];
		}
	}
}


extern "C" void host_code (double *h_ppgc, double *h_ppgk, double *h_wgtfac, double *h_ppuv, int L, int M, int N) {
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpy (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppgk;
	hipMalloc (&ppgk, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgk\n");
	hipMemcpy (ppgk, h_ppgk, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *wgtfac;
	hipMalloc (&wgtfac, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for wgtfac\n");
	hipMemcpy (wgtfac, h_wgtfac, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpy (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice);

	dim3 blockconfig_1 (bx, by, bz); 
	dim3 gridconfig_1 (ceil (N, min(bx1, bx-bx1)), ceil (M, blockconfig_1.y), ceil (L, blockconfig_1.z));

	hipEventRecord(start);
	fusion <<<gridconfig_1, blockconfig_1>>> (ppgc, ppgk, wgtfac, ppuv, L, M, N);;
	hipEventRecord(stop);
	hipMemcpy (h_ppgc, ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);
	hipMemcpy (h_ppgk, ppgk, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost);

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	printf("BlockDim1 (%d, %d, %d)\n", blockconfig_1.x, blockconfig_1.y, blockconfig_1.y);
	printf("GridDim (%d, %d, %d)\n", gridconfig_1.x, gridconfig_1.y, gridconfig_1.y);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}
#include <stdio.h>
#include "hip/hip_runtime.h"
#define max(x,y)    ((x) > (y) ? (x) : (y))
#define min(x,y)    ((x) < (y) ? (x) : (y))
#define ceil(a,b)   ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

#define LL 320
#define MM 320
#define NN 320

#define bx 16
#define by 8
#define bz 8

void check_error (const char* message) {
	hipError_t error = hipGetLastError ();
	if (error != hipSuccess) {
		printf ("CUDA error : %s, %s\n", message, hipGetErrorString (error));
		exit(-1);
	}
}

__global__ void fastwave1 (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-1) && i >= i0 && i <= min(i0+blockDim.x-1, N-2)) {
		ppgu[k][j][i] = (ppuv[k][j][i+1] - ppuv[k][j][i]) + (ppgc[k][j][i+1] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j][i+1] + hhl[k][j][i+1])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j][i+1] + hhl[k][j][i+1]));
	}
}

__global__ void fastwave2 (double * __restrict__ d_ppgu, double * __restrict__ d_ppuv, double * __restrict__ d_ppgc, double * __restrict__ d_hhl, int L, int M, int N) {
	//Determining the block's indices
	int i0 = (int)(blockIdx.x)*((int)blockDim.x);
	int i = i0 + (int)(threadIdx.x);
	int j0 = (int)(blockIdx.y)*((int)blockDim.y);
	int j = j0 + (int)(threadIdx.y);
	int k0 = (int)(blockIdx.z)*((int)blockDim.z);
	int k = k0 + (int)(threadIdx.z);

	double (*ppgu)[MM][NN] = (double (*)[MM][NN]) d_ppgu;
	double (*ppuv)[MM][NN] = (double (*)[MM][NN]) d_ppuv;
	double (*ppgc)[MM][NN] = (double (*)[MM][NN]) d_ppgc;
	double (*hhl)[MM][NN] = (double (*)[MM][NN]) d_hhl;

	//Array and variable declarations

	if (k >= k0 && k <= min(k0+blockDim.z-1, L-2) && j >= j0 && j <= min(j0+blockDim.y-1, M-2) && i >= i0 && i <= min(i0+blockDim.x-1, N-1)) {
		ppgu[k][j][i] = (ppuv[k][j+1][i] - ppuv[k][j][i]) + (ppgc[k][j+1][i] + ppgc[k][j][i]) * 0.5 * ((hhl[k+1][j][i] + hhl[k][j][i]) - (hhl[k+1][j+1][i] + hhl[k][j+1][i])) / ((hhl[k+1][j][i] - hhl[k][j][i]) + (hhl[k+1][j+1][i] + hhl[k][j+1][i]));
	}
}


extern "C" void host_code (double *h_ppgu0, double *h_ppgu1,double *h_ppuv, double *h_ppgc, double *h_hhl, int L, int M, int N) {
	
	hipEvent_t start, stop; 
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipStream_t s1, s2;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	
	double *ppgu0;
	hipMalloc (&ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu0\n");
	//cudaMemcpy (ppgu0, h_ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), cudaMemcpyHostToDevice);
	double *ppgu1;
	hipMalloc (&ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgu1\n");
	//cudaMemcpy (ppgu1, h_ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), cudaMemcpyHostToDevice);
	double *ppuv;
	hipMalloc (&ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppuv\n");
	hipMemcpyAsync (ppuv, h_ppuv, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);
	double *ppgc;
	hipMalloc (&ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for ppgc\n");
	hipMemcpyAsync (ppgc, h_ppgc, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);
	double *hhl;
	hipMalloc (&hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0));
	check_error ("Failed to allocate device memory for hhl\n");
	hipMemcpyAsync (hhl, h_hhl, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyHostToDevice, s1);

	dim3 blockconfig_1 (bx, by, bz);
	dim3 gridconfig_1 (ceil (N - 1 - 0 + 1, blockconfig_1.x), ceil (M - 1 - 0 + 1, blockconfig_1.y), ceil (L - 1 - 0 + 1, blockconfig_1.z));
	hipEventRecord(start);
	fastwave1 <<<gridconfig_1, blockconfig_1, 0, s1>>> (ppgu0, ppuv, ppgc, hhl, L, M, N);
	fastwave2 <<<gridconfig_1, blockconfig_1, 0, s2>>> (ppgu1, ppuv, ppgc, hhl, L, M, N);
	hipEventRecord(stop);
	hipMemcpyAsync (h_ppgu0, ppgu0, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost, s1);
	hipMemcpyAsync (h_ppgu1, ppgu1, sizeof(double )*(L - 0)*(M - 0)*(N - 0), hipMemcpyDeviceToHost, s2);
	
	hipStreamDestroy(s1);
	hipStreamDestroy(s2);
	
	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Kernel time %fs\n", time);
	hipEventDestroy(start);
	hipEventDestroy(stop);
}